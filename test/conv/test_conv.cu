#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <hipblas.h>
#include <hiprand.h>
#include <string>

#include "../../tensor.cu"
#include "../../layers/conv.cu"

int main(){
    hipblasHandle_t handle;
    hiprandGenerator_t prng;
    hipblasCreate(&handle);
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    std::ofstream f_x("X.txt"), f_w("W.txt"), f_y("Y.txt");
    std::ofstream f_dx("Dx.txt"), f_dw("Dw.txt"), f_dy("Dy.txt");

    int n=3, c_in=5, h=7, w=9;
    int c_out = 3;

    tensor<float> *X = new tensor<float>(std::vector<int>{n, c_in, h, w}, "gpu");
    hiprandGenerateUniform(prng, X->data, n*c_in*h*w);
    
    conv<float> conv_ = conv<float>(c_in, c_out, handle, prng);
    tensor<float> *Y = conv_.forward(X);

    tensor<float> *dY = new tensor<float>(Y->shape, "gpu");
    hiprandGenerateUniform(prng, dY->data, n*c_out*h*w);

    std::vector<tensor<float>*> dxdw = conv_.backward(dY);

    tensor<float> *dX = dxdw[0], *dW = dxdw[1];

    tensor<float> *W = conv_.get_w();

    X->f_print(f_x);
    W->f_print(f_w);
    Y->f_print(f_y);
    dX->f_print(f_dx);
    dW->f_print(f_dw);
    dY->f_print(f_dy);

    delete X;
    delete dY;
    hipblasDestroy(handle);
    hiprandDestroyGenerator(prng);
}
