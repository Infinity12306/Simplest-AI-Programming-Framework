#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <random>
#include <hipblas.h>
#include <hiprand.h>

#include "../../tensor.cu"
#include "../../layers/cross_entropy.cu"

int main(){
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    std::ofstream f_x("X.txt"), f_label("labels.txt"), f_y("loss.txt");
    std::ofstream f_dx("Dx.txt");

    int n = 5, c = 7;

    tensor<float> *X = new tensor<float>(std::vector<int>{n, c}, "gpu");
    hiprandGenerateUniform(prng, X->data, X->size / sizeof(float));

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dist(0, c-1);
    tensor<int> *labels = new tensor<int>(std::vector<int>{n}, "cpu");
    for (int i=0; i<n; i++)
        labels->data[i] = dist(gen);
    labels->gpu();

    cross_entropy<float> l_ce = cross_entropy<float>();

    float loss = l_ce.forward(X, labels);
    f_y << loss;

    tensor<float> *dX = l_ce.backward();

    X->f_print(f_x);
    
    dX->f_print(f_dx);
    labels->f_print(f_label);

    delete X;
    X = nullptr;
    delete labels;
    labels = nullptr;
    hiprandDestroyGenerator(prng);
}
